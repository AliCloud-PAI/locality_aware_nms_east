#include "hip/hip_runtime.h"
// Author: Yiwu Yao
// Date: 2019-06-05
// Description: modified for polygon-nms: the shape of input array is N*9,
// containing coordinates of the 4 vertices and score. The NMS from MXnet
// MultiBoxDetection is used.

#include "rotate_gpu_nms.hpp"
#include <vector>
#include <iostream>
#include <cmath>

#define CUDA_CHECK(condition) \
  /* Code block avoids redefinition of hipError_t error */ \
  do { \
    hipError_t error = condition; \
    if (error != hipSuccess) { \
      std::cout << hipGetErrorString(error) << std::endl; \
    } \
  } while (0)

int const threadsPerBlock = 1024;

__device__ inline float sqr_d(float x) { return x * x; }

__device__ inline float trangle_area(float * a, float * b, float * c) {
  return ((a[0] - c[0]) * (b[1] - c[1]) - (a[1] - c[1]) * (b[0] - c[0]))/2.0;
}

__device__ inline float area(float * int_pts, int num_of_inter) {

  float area = 0.0;
  for(int i = 0;i < num_of_inter - 2;i++) {
    area += fabs(trangle_area(int_pts, int_pts + 2 * i + 2, int_pts + 2 * i + 4));
  }
  return area;
}

__device__ inline float trangle_area_rect(const float * a, const float * b, const float * c) {
  return ((a[0] - c[0]) * (b[1] - c[1]) - (a[1] - c[1]) * (b[0] - c[0]))/2.0;
}

__device__ inline float area_rect(const float * int_pts, int num_of_inter) {

  float area = 0.0;
  for(int i = 0;i < num_of_inter - 2;i++) {
    area += fabs(trangle_area_rect(int_pts, int_pts + 2 * i + 2, int_pts + 2 * i + 4));
  }
  return area;
}

__device__ inline void reorder_pts(float * int_pts, int num_of_inter) {

  if(num_of_inter > 0) {

    float center[2];
    center[0] = 0.0;
    center[1] = 0.0;

    for(int i = 0;i < num_of_inter;i++) {
      center[0] += int_pts[2 * i];
      center[1] += int_pts[2 * i + 1];
    }
    center[0] /= num_of_inter;
    center[1] /= num_of_inter;

    float vs[16];
    float v[2];
    float d;
    for(int i = 0;i < num_of_inter;i++) {
      v[0] = int_pts[2 * i]-center[0];
      v[1] = int_pts[2 * i + 1]-center[1];
      d = sqrt(v[0] * v[0] + v[1] * v[1]);
      v[0] = v[0] / d;
      v[1] = v[1] / d;
      if(v[1] < 0) {
        v[0]= - 2 - v[0];
      }
      vs[i] = v[0];
    }

    float temp,tx,ty;
    int j;
    for(int i=1;i<num_of_inter;++i){
      if(vs[i-1]>vs[i]){
        temp = vs[i];
        tx = int_pts[2*i];
        ty = int_pts[2*i+1];
        j=i;
        while(j>0&&vs[j-1]>temp){
          vs[j] = vs[j-1];
          int_pts[j*2] = int_pts[j*2-2];
          int_pts[j*2+1] = int_pts[j*2-1];
          j--;
        }
        vs[j] = temp;
        int_pts[j*2] = tx;
        int_pts[j*2+1] = ty;
      }
    }
  }

}
__device__ inline bool inter2line(float * pts1, float *pts2, int i, int j, float * temp_pts) {

  float a[2];
  float b[2];
  float c[2];
  float d[2];

  float area_abc, area_abd, area_cda, area_cdb;

  a[0] = pts1[2 * i];
  a[1] = pts1[2 * i + 1];

  b[0] = pts1[2 * ((i + 1) % 4)];
  b[1] = pts1[2 * ((i + 1) % 4) + 1];

  c[0] = pts2[2 * j];
  c[1] = pts2[2 * j + 1];

  d[0] = pts2[2 * ((j + 1) % 4)];
  d[1] = pts2[2 * ((j + 1) % 4) + 1];

  area_abc = trangle_area(a, b, c);
  area_abd = trangle_area(a, b, d);

  if(area_abc * area_abd >= 0) {
    return false;
  }

  area_cda = trangle_area(c, d, a);
  area_cdb = area_cda + area_abc - area_abd;

  if (area_cda * area_cdb >= 0) {
    return false;
  }
  float t = area_cda / (area_abd - area_abc);

  float dx = t * (b[0] - a[0]);
  float dy = t * (b[1] - a[1]);
  temp_pts[0] = a[0] + dx;
  temp_pts[1] = a[1] + dy;

  return true;
}

__device__ inline bool in_rect(float pt_x, float pt_y, float * pts) {

  float ab[2];
  float ad[2];
  float ap[2];

  float abab;
  float abap;
  float adad;
  float adap;

  ab[0] = pts[2] - pts[0];
  ab[1] = pts[3] - pts[1];

  ad[0] = pts[6] - pts[0];
  ad[1] = pts[7] - pts[1];

  ap[0] = pt_x - pts[0];
  ap[1] = pt_y - pts[1];

  abab = ab[0] * ab[0] + ab[1] * ab[1];
  abap = ab[0] * ap[0] + ab[1] * ap[1];
  adad = ad[0] * ad[0] + ad[1] * ad[1];
  adap = ad[0] * ap[0] + ad[1] * ap[1];

  return abab >= abap and abap >= 0 and adad >= adap and adap >= 0;
}

__device__ inline int inter_pts(float * pts1, float * pts2, float * int_pts) {

  int num_of_inter = 0;

  for(int i = 0;i < 4;i++) {
    if(in_rect(pts1[2 * i], pts1[2 * i + 1], pts2)) {
      int_pts[num_of_inter * 2] = pts1[2 * i];
      int_pts[num_of_inter * 2 + 1] = pts1[2 * i + 1];
      num_of_inter++;
    }
     if(in_rect(pts2[2 * i], pts2[2 * i + 1], pts1)) {
      int_pts[num_of_inter * 2] = pts2[2 * i];
      int_pts[num_of_inter * 2 + 1] = pts2[2 * i + 1];
      num_of_inter++;
    }
  }

  float temp_pts[2];

  for(int i = 0;i < 4;i++) {
    for(int j = 0;j < 4;j++) {
      bool has_pts = inter2line(pts1, pts2, i, j, temp_pts);
      if(has_pts) {
        int_pts[num_of_inter * 2] = temp_pts[0];
        int_pts[num_of_inter * 2 + 1] = temp_pts[1];
        num_of_inter++;
      }
    }
  }


  return num_of_inter;
}

__device__ inline void convert_region(float * pts , float const * const region) {

  for(int i = 0;i < 4;i++) {
    pts[7 - 2 * i - 1] = region[2*i];
    pts[7 - 2 * i] = region[2*i+1];
  }
}


__device__ inline float inter(float const * const region1, float const * const region2) {

  float pts1[8], pts2[8];
  float int_pts[16];
  int num_of_inter;

  convert_region(pts1, region1);
  convert_region(pts2, region2);

  num_of_inter = inter_pts(pts1, pts2, int_pts);
  reorder_pts(int_pts, num_of_inter);

  return area(int_pts, num_of_inter);

}

__device__ inline float devRotateIoU(float const * const region1, float const * const region2) {

  float area1 = area_rect(region1, 4);
  float area2 = area_rect(region2, 4);
  float area_inter = inter(region1, region2);

  return area_inter / (area1 + area2 - area_inter);

}

__global__ void rotate_nms_kernel(const int n_boxes, const float nms_overlap_thresh,
                           const float *dev_boxes, float *out) {
  int index = threadIdx.x;

  // define dynamic shared memory to cache all boxes
  /*
  extern __shared__ float block_boxes[]; // 4 vertices and s

  for (int i = index; i < n_boxes; i += blockDim.x) {
    for (int j = 0; j < 9; j++) {
      block_boxes[i * 9 + j] = dev_boxes[i * 9 + j];
    }
  }
  __syncthreads();
  */
  for (int i = index; i < n_boxes; i += blockDim.x) {
    // store the first bbox
    out[i * 11] = 1; // 1: valid, -1: invalid
    out[i * 11 + 10] = -1;
    for (int j = 0; j < 9; j++) {
      out[i * 11 + j + 1] = dev_boxes[i * 9 + j];
    }
  }
  __syncthreads();

  // apply NMS, from MXNet MultiBoxDetection
  for (int compare_pos = 0; compare_pos < n_boxes; ++compare_pos) {
    float compare_id = out[compare_pos * 11];
    if (compare_id < 0) continue;  // not a valid positive detection, skip
    float *compare_loc_ptr = out + compare_pos * 11 + 1;
    for (int i = compare_pos + index + 1; i < n_boxes; i += blockDim.x) {
      float class_id = out[i * 11];
      if (class_id < 0) continue;
      if (devRotateIoU(compare_loc_ptr, out + i * 11 + 1) > nms_overlap_thresh) {
        out[i * 11] = -1;
        out[i * 11 + 10] = static_cast<float>(compare_pos);
      }
    }
    __syncthreads();
  }

  // post merge
  for (int i = index; i < n_boxes; i += blockDim.x) {
    int ref = i * 11; // the reference and update one
    if (out[ref] > 0) { // if valid
      //int count = 0;
      float score_t = out[ref + 9];
      for (int k=1; k < (n_boxes-i); k+=1) {
        int p_given = (i+k)*11; // the followed one
        int invalid_i = static_cast<int>(out[p_given+10]); // if invalid, corresponding to i
        if ((out[p_given] < 0) && (invalid_i == i)){
          //if (devRotateIoU(out + ref + 1, out + p_given + 1) > nms_overlap_thresh) {
            // merge
            /*
            if ((out[p_given+7] < out[ref+7]) ) {
                out[ref+7] = out[p_given+7];
                out[ref+8] = out[p_given+8];
            }
            if ((out[p_given+1] < out[ref+1]) ) {
                out[ref+1] = out[p_given+1];
                out[ref+2] = out[p_given+2];
            }

            if ((out[p_given+5] >= out[ref+5]) ) {
                out[ref+5] = out[p_given+5];
                out[ref+6] = out[p_given+6];
            }
            if ((out[p_given+3] >= out[ref+3]) ) {
                out[ref+3] = out[p_given+3];
                out[ref+4] = out[p_given+4];
            }
            */
            float p_s = out[p_given + 9];
    				for (int v = 0; v < 4; v++) {
              out[ref+v*2+1] = (out[ref+v*2+1]*score_t + out[p_given+v*2+1]*p_s) / (score_t + p_s);
              out[ref+v*2+2] = (out[ref+v*2+2]*score_t + out[p_given+v*2+2]*p_s) / (score_t + p_s);
            }
            score_t += p_s; // update the total score
            //if (count>1024) break;
            //count++;
          //}
        }
      }
      out[ref + 9] = score_t;
    }
  }
  __syncthreads();
}

void _set_device(int device_id) {
  int current_device;
  CUDA_CHECK(hipGetDevice(&current_device));
  if (current_device == device_id) {
    return;
  }
  // The call to hipSetDevice must come before any calls to Get, which
  // may perform initialization using the GPU.
  CUDA_CHECK(hipSetDevice(device_id));
}

// Host routine
void _rotate_nms(float *nms_out_host, int *num_out, const float* boxes_host, const int boxes_num, const int boxes_dim,
                 float nms_overlap_thresh, int device_id) {
  _set_device(device_id);

  float* boxes_dev = NULL;
  float* out_dev = NULL;

  CUDA_CHECK(hipMalloc(&boxes_dev,
                        boxes_num * boxes_dim * sizeof(float)));
  CUDA_CHECK(hipMemcpy(boxes_dev,
                        boxes_host,
                        boxes_num * boxes_dim * sizeof(float),
                        hipMemcpyHostToDevice));

  CUDA_CHECK(hipMalloc(&out_dev,
                        boxes_num * (boxes_dim+2) * sizeof(float)));

  dim3 threads(threadsPerBlock);
  rotate_nms_kernel<<<1, threads>>>(boxes_num,
                                  nms_overlap_thresh,
                                  boxes_dev,
                                  out_dev);

  // dynamic array allocation
  float (* out_host)[11] = new float[boxes_num][11]; // 11 = boxes_dim+2
  CUDA_CHECK(hipMemcpy(&out_host[0][0],
                        out_dev,
                        sizeof(float) * boxes_num * (boxes_dim+2),
                        hipMemcpyDeviceToHost));

  // delete the invalid bbox
  int num_to_keep = 0;
  for (int i = 0; i < boxes_num; i++) {
    int id = int(out_host[i][0]);
    if (id > 0) {
      for (int j = 0; j < boxes_dim; j++) {
        nms_out_host[num_to_keep * boxes_dim + j] = out_host[i][j+1];
      }
      num_to_keep++;
    }
  }
  *num_out = num_to_keep;

  // clean up
  CUDA_CHECK(hipFree(boxes_dev));
  CUDA_CHECK(hipFree(out_dev));
  delete[] out_host;
}
